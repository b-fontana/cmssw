#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "RecoLocalCalo/HGCalRecProducers/plugins/KernelManagerHGCalRecHit.h"
#include "RecoLocalCalo/HGCalRecProducers/plugins/HGCalRecHitKernelImpl.cuh"

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(HGCUncalibratedRecHitSoA* h_uncalibSoA,
                                                   HGCUncalibratedRecHitSoA* d_uncalibSoA,
                                                   HGCRecHitSoA* d_calibSoA)
    : h_uncalibSoA_(h_uncalibSoA), d_uncalibSoA_(d_uncalibSoA), d_calibSoA_(d_calibSoA) {
  nhits_ = h_uncalibSoA_->nhits_;
  stride_ = h_uncalibSoA_->stride_;
  ::nb_rechits_ = (stride_ + ::nt_rechits_.x - 1) / ::nt_rechits_.x;
  nbytes_device_ = d_uncalibSoA_->nbytes_ * stride_;
}

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(HGCRecHitSoA* h_calibSoA, HGCRecHitSoA* d_calibSoA)
    : h_calibSoA_(h_calibSoA), d_calibSoA_(d_calibSoA) {
  nhits_ = h_calibSoA_->nhits_;
  stride_ = h_calibSoA_->stride_;
  ::nb_rechits_ = (stride_ + ::nt_rechits_.x - 1) / ::nt_rechits_.x;
  nbytes_host_ = h_calibSoA_->nbytes_ * stride_;
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit() {}

void KernelManagerHGCalRecHit::transfer_soa_to_device_(const hipStream_t& stream) {
  cudaCheck(hipMemcpyAsync(
      d_uncalibSoA_->amplitude_, h_uncalibSoA_->amplitude_, nbytes_device_, hipMemcpyHostToDevice, stream));
  cudaCheck(hipGetLastError());
}

void KernelManagerHGCalRecHit::transfer_soa_to_host(const hipStream_t& stream) {
  cudaCheck(hipMemcpyAsync(h_calibSoA_->energy_, d_calibSoA_->energy_, nbytes_host_, hipMemcpyDeviceToHost, stream));
  cudaCheck(hipGetLastError());
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData>* kcdata,
                                           const hipStream_t& stream) {
  transfer_soa_to_device_(stream);
  ee_to_rechit<<<::nb_rechits_, ::nt_rechits_, 0, stream>>>(*d_calibSoA_, *d_uncalibSoA_, kcdata->data_, nhits_);
  cudaCheck(hipGetLastError());
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData>* kcdata,
                                           const hipStream_t& stream) {
  transfer_soa_to_device_(stream);
  hef_to_rechit<<<::nb_rechits_, ::nt_rechits_, 0, stream>>>(*d_calibSoA_, *d_uncalibSoA_, kcdata->data_, nhits_);
  cudaCheck(hipGetLastError());
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData>* kcdata,
                                           const hipStream_t& stream) {
  transfer_soa_to_device_(stream);
  heb_to_rechit<<<::nb_rechits_, ::nt_rechits_, 0, stream>>>(*d_calibSoA_, *d_uncalibSoA_, kcdata->data_, nhits_);
  cudaCheck(hipGetLastError());
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output() { return h_calibSoA_; }
