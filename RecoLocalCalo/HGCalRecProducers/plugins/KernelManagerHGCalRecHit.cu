#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "RecoLocalCalo/HGCalRecProducers/plugins/KernelManagerHGCalRecHit.h"
#include "RecoLocalCalo/HGCalRecProducers/plugins/HGCalRecHitKernelImpl.cuh"

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit()
{
  ::nb_rechits_ = (data_->nhits_ + ::nt_rechits_.x - 1) / ::nt_rechits_.x;
}

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelModifiableData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> *data):
  data_(data)
{
  ::nb_rechits_ = (data_->nhits_ + ::nt_rechits_.x - 1) / ::nt_rechits_.x;
  nbytes_host_ = (data_->h_out_)->nbytes_ * data_->stride_;
  nbytes_device_ = (data_->d_1_)->nbytes_ * data_->stride_;
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::transfer_soas_to_device_(const hipStream_t& stream)
{
  cudaCheck( hipMemcpyAsync((data_->d_1_)->amplitude_, (data_->h_in_)->amplitude_, nbytes_device_, hipMemcpyHostToDevice, stream) );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::transfer_soa_to_host_and_synchronize_(const hipStream_t& stream)
{
  cudaCheck( hipMemcpyAsync((data_->h_out_)->energy_, (data_->d_out_)->energy_, nbytes_host_, hipMemcpyDeviceToHost, stream) );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::reuse_device_pointers_()
{
  std::swap(data_->d_1_, data_->d_2_); 
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *kcdata, const hipStream_t& stream)
{
  transfer_soas_to_device_( stream );
  cudaCheck( hipGetLastError() );
    
  /*
  ee_step1<<<::nb_rechits_, ::nt_rechits_, 0, stream>>>( *(data_->d_2_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  after_();
  reuse_device_pointers_();
  */

  ee_to_rechit<<<::nb_rechits_, ::nt_rechits_, 0, stream>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  cudaCheck( hipGetLastError() );

  transfer_soa_to_host_and_synchronize_( stream );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *kcdata, const hipStream_t& stream)
{
  transfer_soas_to_device_( stream );
  cudaCheck( hipGetLastError() );

  /*
  hef_step1<<<::nb_rechits_, ::nt_rechits_>>>( *(data_->d_2), *(data_->d_1_), d_kcdata->data, data_->nhits_);
  cudaCheck( hipGetLastError() );
  reuse_device_pointers_();
  */

  hef_to_rechit<<<::nb_rechits_, ::nt_rechits_, 0, stream>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  cudaCheck( hipGetLastError() );

  transfer_soa_to_host_and_synchronize_( stream );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *kcdata, const hipStream_t& stream)
{
  transfer_soas_to_device_( stream );
  cudaCheck( hipGetLastError() );

  /*
  heb_step1<<<::nb_rechits_, nt_rechits_>>>( *(data_->d_2_), *(data_->d_1_), d_kcdata->data_, data_->nhits_);
  cudaCheck( hipGetLastError() );
  reuse_device_pointers_();
  */

  heb_to_rechit<<<::nb_rechits_, ::nt_rechits_, 0, stream>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  cudaCheck( hipGetLastError() );

  transfer_soa_to_host_and_synchronize_( stream );
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_->h_out_;
}
