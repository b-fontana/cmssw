#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "KernelManager.h"
#include "HGCalRecHitKernelImpl.cuh"

dim3 nblocks_;
constexpr dim3 nthreads_(256); //some kernels will potentially not allocate shared memory properly with a lower number

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelModifiableData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> *data):
  data_(data)
{
  nblocks_ = (data_->nhits + nthreads_.x - 1) / nthreads_.x;
  nbytes_host_ = (data_->h_out)->nbytes * data_->stride;
  nbytes_device_ = (data_->d_1)->nbytes * data_->stride;
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_()
{
  cudaCheck( hipMemcpyAsync((data_->d_1)->amplitude, (data_->h_in)->amplitude, nbytes_device_, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGCeeUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.hgcEE_fCPerMIP_, h_kcdata->data.hgcEE_fCPerMIP_, h_kcdata->data.nbytes, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChefUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.hgcHEF_fCPerMIP_, h_kcdata->data.hgcHEF_fCPerMIP_, h_kcdata->data.nbytes, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::assign_and_transfer_to_device_(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChebUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.weights_, h_kcdata->data.weights_, h_kcdata->data.nbytes, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::transfer_to_host_and_synchronize_()
{
  cudaCheck( hipMemcpyAsync((data_->h_out)->energy, (data_->d_out)->energy, nbytes_host_, hipMemcpyDeviceToHost) );
  after_();
}

void KernelManagerHGCalRecHit::reuse_device_pointers_()
{
  std::swap(data_->d_1, data_->d_2); 
  after_();
}

int KernelManagerHGCalRecHit::get_shared_memory_size_(const int& nd, const int& nf, const int& nu, const int& ni, const int& nb) {
  int dmem = nd*sizeof(double);
  int fmem = nf*sizeof(float);
  int umem = nu*sizeof(uint32_t);
  int imem = ni*sizeof(int);
  int bmem = nb*sizeof(bool);
  return dmem + fmem + umem + imem + bmem;
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGCeeUncalibratedRecHitConstantData> *d_kcdata)
{
  assign_and_transfer_to_device_(h_kcdata, d_kcdata);
  assign_and_transfer_to_device_();

  printf("%d blocks being launched with %d threads (%d in total) for %d ee hits.\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x, data_->nhits);
  int nbytes_shared = get_shared_memory_size_(h_kcdata->data.ndelem, h_kcdata->data.nfelem, h_kcdata->data.nuelem, h_kcdata->data.nielem, h_kcdata->data.nbelem);

  /*
  ee_step1<<<nblocks_, nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();
  reuse_device_pointers_();
  */

  ee_to_rechit<<<nblocks_, nthreads_, nbytes_shared>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();

  transfer_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChefUncalibratedRecHitConstantData> *d_kcdata)
{
  assign_and_transfer_to_device_(h_kcdata, d_kcdata);
  assign_and_transfer_to_device_();

  printf("%d blocks being launched with %d threads (%d in total) for %d hef hits.\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x, data_->nhits);
  int nbytes_shared = get_shared_memory_size_(h_kcdata->data.ndelem, h_kcdata->data.nfelem, h_kcdata->data.nuelem, h_kcdata->data.nielem, h_kcdata->data.nbelem);

  /*
  hef_step1<<<nblocks_,nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits);
  after_();
  reuse_device_pointers_();
  */

  hef_to_rechit<<<nblocks_,nthreads_, nbytes_shared>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();
  transfer_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChebUncalibratedRecHitConstantData> *d_kcdata)
{
  assign_and_transfer_to_device_(h_kcdata, d_kcdata);
  assign_and_transfer_to_device_();

  printf("%d blocks being launched with %d threads (%d in total) for %d heb hits.\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x, data_->nhits);
  int nbytes_shared = get_shared_memory_size_(h_kcdata->data.ndelem, h_kcdata->data.nfelem, h_kcdata->data.nuelem, h_kcdata->data.nielem, h_kcdata->data.nbelem);

  /*
  heb_step1<<<nblocks_, nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits);
  after_();
  reuse_device_pointers_();
  */

  heb_to_rechit<<<nblocks_, nthreads_, nbytes_shared>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();

  transfer_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::after_() {
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_->h_out;
}
