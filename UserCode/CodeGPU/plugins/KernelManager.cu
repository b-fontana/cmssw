#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "KernelManager.h"
#include "HGCalRecHitKernelImpl.cuh"

dim3 nblocks_;
constexpr dim3 nthreads_(256); //some kernels will potentially not allocate shared memory properly with a lower number

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelModifiableData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> *data):
  data_(data)
{
  nblocks_ = (data_->nhits + nthreads_.x - 1) / nthreads_.x;
  nbytes_host_ = (data_->h_out)->nbytes * data_->stride;
  nbytes_device_ = (data_->d_1)->nbytes * data_->stride;
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::transfer_soas_to_device_()
{
  cudaCheck( hipMemcpyAsync((data_->d_1)->amplitude, (data_->h_in)->amplitude, nbytes_device_, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::transfer_constants_to_device_(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGCeeUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.hgcEE_fCPerMIP_, h_kcdata->data.hgcEE_fCPerMIP_, h_kcdata->data.nbytes_, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::transfer_constants_to_device_(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChefUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.hgcHEF_fCPerMIP_, h_kcdata->data.hgcHEF_fCPerMIP_, h_kcdata->data.nbytes_, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::transfer_constants_to_device_(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChebUncalibratedRecHitConstantData> *d_kcdata)
{
  cudaCheck( hipMemcpyAsync( d_kcdata->data.weights_, h_kcdata->data.weights_, h_kcdata->data.nbytes_, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::transfer_soa_to_host_and_synchronize_()
{
  cudaCheck( hipMemcpyAsync((data_->h_out)->energy, (data_->d_out)->energy, nbytes_host_, hipMemcpyDeviceToHost) );
  after_();
}

void KernelManagerHGCalRecHit::reuse_device_pointers_()
{
  std::swap(data_->d_1, data_->d_2); 
  after_();
}

int KernelManagerHGCalRecHit::get_shared_memory_size_(const int& nd, const int& nf, const int& nu, const int& ni) {
  int dmem = nd*sizeof(double);
  int fmem = nf*sizeof(float);
  int umem = nu*sizeof(uint32_t);
  int imem = ni*sizeof(int);
  return dmem + fmem + umem + imem;
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGCeeUncalibratedRecHitConstantData> *d_kcdata)
{
  transfer_constants_to_device_(h_kcdata, d_kcdata);
  transfer_soas_to_device_();

  printf("%d blocks being launched with %d threads (%d in total) for %d ee hits.\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x, data_->nhits);
  int nbytes_shared = get_shared_memory_size_(h_kcdata->data.ndelem_, h_kcdata->data.nfelem_, h_kcdata->data.nuelem_, h_kcdata->data.nielem_);

  /*
  ee_step1<<<nblocks_, nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();
  reuse_device_pointers_();
  */

  ee_to_rechit<<<nblocks_, nthreads_, nbytes_shared>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();

  transfer_soa_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChefUncalibratedRecHitConstantData> *d_kcdata)
{
  transfer_constants_to_device_(h_kcdata, d_kcdata);
  transfer_soas_to_device_();

  printf("%d blocks being launched with %d threads (%d in total) for %d hef hits.\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x, data_->nhits);
  int nbytes_shared = get_shared_memory_size_(h_kcdata->data.ndelem_, h_kcdata->data.nfelem_, h_kcdata->data.nuelem_, h_kcdata->data.nielem_);

  /*
  hef_step1<<<nblocks_,nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits);
  after_();
  reuse_device_pointers_();
  */

  hef_to_rechit<<<nblocks_,nthreads_, nbytes_shared>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();
  transfer_soa_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *h_kcdata, KernelConstantData<HGChebUncalibratedRecHitConstantData> *d_kcdata)
{
  transfer_constants_to_device_(h_kcdata, d_kcdata);
  transfer_soas_to_device_();

  printf("%d blocks being launched with %d threads (%d in total) for %d heb hits.\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x, data_->nhits);
  int nbytes_shared = get_shared_memory_size_(h_kcdata->data.ndelem_, h_kcdata->data.nfelem_, h_kcdata->data.nuelem_, h_kcdata->data.nielem_);

  /*
  heb_step1<<<nblocks_, nthreads_>>>( *(data_->d_2), *(data_->d_1), d_kcdata->data, data_->nhits);
  after_();
  reuse_device_pointers_();
  */

  heb_to_rechit<<<nblocks_, nthreads_, nbytes_shared>>>( *(data_->d_out), *(data_->d_1), d_kcdata->data, data_->nhits );
  after_();

  transfer_soa_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::after_() {
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_->h_out;
}
