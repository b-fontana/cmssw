#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "RecoLocalCalo/HGCalRecProducers/plugins/KernelManagerHGCalRecHit.h"
#include "HGCalRecHitKernelImpl.cuh"

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelModifiableData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> *data):
  data_(data)
{
  ::nblocks_ = (data_->nhits_ + ::nthreads_.x - 1) / ::nthreads_.x;
  nbytes_host_ = (data_->h_out_)->nbytes_ * data_->stride_;
  nbytes_device_ = (data_->d_1_)->nbytes_ * data_->stride_;

  printf("NUMBERS: %d - %d - %d\n", nblocks_.x, nthreads_.x, nblocks_.x*nthreads_.x);
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::transfer_soas_to_device_()
{
  cudaCheck( hipMemcpyAsync((data_->d_1_)->amplitude_, (data_->h_in_)->amplitude_, nbytes_device_, hipMemcpyHostToDevice) );
  after_();
}

void KernelManagerHGCalRecHit::transfer_soa_to_host_and_synchronize_()
{
  cudaCheck( hipMemcpyAsync((data_->h_out_)->energy_, (data_->d_out_)->energy_, nbytes_host_, hipMemcpyDeviceToHost) );
  after_();
}

void KernelManagerHGCalRecHit::reuse_device_pointers_()
{
  std::swap(data_->d_1_, data_->d_2_); 
  after_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *kcdata)
{
  transfer_soas_to_device_();
  /*
  ee_step1<<<::nblocks_, ::nthreads_>>>( *(data_->d_2_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  after_();
  reuse_device_pointers_();
  */

  ee_to_rechit<<<::nblocks_, ::nthreads_>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  after_();
  transfer_soa_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *kcdata, const hgcal_conditions::HeterogeneousHEFConditionsESProduct* d_conds)
{
  transfer_soas_to_device_();

  fill_positions_from_detids<<<::nblocks_,::nthreads_>>>(d_conds);
  after_();
  
  print_positions_from_detids<<<::nblocks_,::nthreads_>>>(d_conds);
  /*
  hef_step1<<<::nblocks_,::nthreads_>>>( *(data_->d_2), *(data_->d_1_), d_kcdata->data, data_->nhits_);
  after_();
  reuse_device_pointers_();
  */

  //hef_to_rechit<<<::nblocks_,::nthreads_>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, d_conds, data_->nhits_ );
  after_();

  transfer_soa_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *kcdata)
{
  transfer_soas_to_device_();

  /*
  heb_step1<<<::nblocks_, ::nthreads_>>>( *(data_->d_2_), *(data_->d_1_), d_kcdata->data_, data_->nhits_);
  after_();
  reuse_device_pointers_();
  */

  heb_to_rechit<<<::nblocks_, ::nthreads_>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  after_();
  transfer_soa_to_host_and_synchronize_();
}

void KernelManagerHGCalRecHit::after_() {
  cudaCheck( hipDeviceSynchronize() );
  cudaCheck( hipGetLastError() );
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_->h_out_;
}
