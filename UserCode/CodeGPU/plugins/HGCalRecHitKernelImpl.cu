#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "DataFormats/HGCRecHit/interface/HGCRecHit.h"
#include "DataFormats/ForwardDetId/interface/HGCalDetId.h"
#include "HGCalRecHitKernelImpl.cuh"
#include "HGCalDetIdTools.h"

__device__ 
double get_weight_from_layer(const int& padding, const int& layer, double*& sd)
{
  return sd[padding + layer];
}

__device__
void make_rechit(unsigned int tid, HGCRecHitSoA& dst_soa, HGCUncalibratedRecHitSoA& src_soa, const bool &heb_flag, 
		 const double &weight, const double &rcorr, const double &cce_correction, const double &sigmaNoiseGeV, float *& sf)
{
  dst_soa.id_[tid] = src_soa.id_[tid];
  dst_soa.energy_[tid] = src_soa.amplitude_[tid] * weight * 0.001f;
  if(!heb_flag)
    dst_soa.energy_[tid] *=  __fdividef(rcorr, cce_correction);
  dst_soa.time_[tid] = src_soa.jitter_[tid];
  dst_soa.flagBits_[tid] |= (0x1 << HGCRecHit::kGood);
  float son = __fdividef( dst_soa.energy_[tid], sigmaNoiseGeV);
  float son_norm = fminf(32.f, son) / 32.f * ((1 << 8)-1);
  long int son_round = lroundf( son_norm );
  dst_soa.son_[tid] = static_cast<uint8_t>( son_round );

  if(heb_flag==0)
    {
      //get time resolution
      float max = fmaxf(son, sf[0]); //this max trick avoids if...elseif...else condition
      float aterm = sf[2];
      float cterm = sf[3];
      dst_soa.timeError_[tid] = sqrt( __fdividef(aterm,max)*__fdividef(aterm,max) + cterm*cterm );
    }
  else
    dst_soa.timeError_[tid] = -1;
}

__device__ 
double get_thickness_correction(const int& padding, const int& type, double *& sd)
{
  return sd[padding + type];
}

__device__
double get_noise(const int& padding, const int& type, double *& sd)
{
  return sd[padding + type - 1];
}

__device__
double get_cce_correction(const int& padding, const int& type, double *& sd)
{
  return sd[padding + type - 1];
}

__device__ 
double get_fCPerMIP(const int& padding, const int& type, double *& sd)
{
  return sd[padding + type - 1];
}

__device__ 
void set_shared_memory(const int& tid, double*& sd, float*& sf, int*& si, const HGCeeUncalibratedRecHitConstantData& cdata, const int& size1, const int& size2, const int& size3, const int& size4, const int& size5)
{
  const int initial_pad = 2;
  if(tid == 0)
    {
      sd[0] = cdata.hgcEE_keV2DIGI_;
      sd[1] = cdata.hgceeUncalib2GeV_;
      for(unsigned int i=initial_pad; i<size1; ++i)
	sd[i] = cdata.hgcEE_fCPerMIP_[i-initial_pad];
      for(unsigned int i=size1; i<size2; ++i)
	sd[i] = cdata.hgcEE_cce_[i-size1];
      for(unsigned int i=size2; i<size3; ++i)
	sd[i] = cdata.hgcEE_noise_fC_[i-size2];  
      for(unsigned int i=size3; i<size4; ++i)
	sd[i] = cdata.rcorr_[i-size3];
      for(unsigned int i=size4; i<size5; ++i)
	sd[i] = cdata.weights_[i-size4];
      sf[0] = (cdata.xmin_ > 0) ? cdata.xmin_ : 0.1;
      sf[1] = cdata.xmax_;
      sf[2] = cdata.aterm_;
      sf[3] = cdata.cterm_;
    }
}

__device__ 
void set_shared_memory(const int& tid, double*& sd, float*& sf, int*& si, const HGChefUncalibratedRecHitConstantData& cdata, const int& size1, const int& size2, const int& size3, const int& size4, const int& size5)
{
  const int initial_pad = 2;
  if(tid == 0)
    {
      sd[0] = cdata.hgcHEF_keV2DIGI_;
      sd[1] = cdata.hgchefUncalib2GeV_;
      for(unsigned int i=initial_pad; i<size1; ++i)
	sd[i] = cdata.hgcHEF_fCPerMIP_[i-initial_pad];
      for(unsigned int i=size1; i<size2; ++i)
	sd[i] = cdata.hgcHEF_cce_[i-size1];
      for(unsigned int i=size2; i<size3; ++i)
	sd[i] = cdata.hgcHEF_noise_fC_[i-size2];  
      for(unsigned int i=size3; i<size4; ++i)
	sd[i] = cdata.rcorr_[i-size3];
      for(unsigned int i=size4; i<size5; ++i)
	sd[i] = cdata.weights_[i-size4];
      sf[0] = (cdata.xmin_ > 0) ? cdata.xmin_ : 0.1;
      sf[1] = cdata.xmax_;
      sf[2] = cdata.aterm_;
      sf[3] = cdata.cterm_;
    }
}

__device__ 
void set_shared_memory(const int& tid, double*& sd, float*& sf, const HGChebUncalibratedRecHitConstantData& cdata, const int& size1)
{
  const int initial_pad = 3;
  if(tid == 0)
    {
      sd[0] = cdata.hgcHEB_keV2DIGI_;
      sd[1] = cdata.hgchebUncalib2GeV_;
      sd[2] = cdata.hgcHEB_noise_MIP_;
      for(unsigned int i=initial_pad; i<size1; ++i)
	sd[i] = cdata.weights_[i-initial_pad];
    }
}

__global__
void ee_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGCeeUncalibratedRecHitConstantData cdata, int length)
{
}

__global__
void hef_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChefUncalibratedRecHitConstantData cdata, int length)
{
}

__global__
void heb_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChebUncalibratedRecHitConstantData cdata, int length)
{
}

__global__
void ee_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGCeeUncalibratedRecHitConstantData cdata, int length)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  HeterogeneousHGCSiliconDetId detid(src_soa.id_[tid]);
  int size1 = cdata.s_hgcEE_fCPerMIP_ + 2;
  int size2 = cdata.s_hgcEE_cce_      + size1;
  int size3 = cdata.s_hgcEE_noise_fC_ + size2;
  int size4 = cdata.s_rcorr_          + size3; 
  int size5 = cdata.s_weights_        + size4; 

  extern __shared__ double s[];
  double   *sd = s;
  float    *sf = (float*)   (sd + cdata.ndelem_);
  int      *si = (int*)     (sf + cdata.nfelem_);
  set_shared_memory(threadIdx.x, sd, sf, si, cdata, size1, size2, size3, size4, size5);
  __syncthreads();

  for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
      double weight = get_weight_from_layer(size4, detid.layer(), sd);
      double rcorr = get_thickness_correction(size3, detid.layer(), sd);
      double noise = get_noise(size2, detid.type(), sd);
      double cce_correction = get_cce_correction(size1, detid.type(), sd);
      double fCPerMIP = get_fCPerMIP(2, detid.type(), sd);
      double sigmaNoiseGeV = 1e-3 * weight * rcorr * __fdividef( noise, fCPerMIP );
      make_rechit(i, dst_soa, src_soa, false, weight, rcorr, cce_correction, sigmaNoiseGeV, sf);
    }
}

__global__
void hef_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChefUncalibratedRecHitConstantData cdata, const hgcal_conditions::HeterogeneousHEFConditionsESProduct* conds, int length)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  HeterogeneousHGCSiliconDetId detid(src_soa.id_[tid]);
  printf("waferTypeL: %d - cellCoarseY: %lf - cellX: %d\n", conds->params.waferTypeL_[0], conds->params.cellCoarseY_[12], detid.cellX());

  int size1 = cdata.s_hgcHEF_fCPerMIP_ + 2;
  int size2 = cdata.s_hgcHEF_cce_      + size1;
  int size3 = cdata.s_hgcHEF_noise_fC_ + size2;
  int size4 = cdata.s_rcorr_           + size3; 
  int size5 = cdata.s_weights_         + size4; 

  extern __shared__ double s[];
  double   *sd = s;
  float    *sf = (float*)   (sd + cdata.ndelem_);
  int      *si = (int*)     (sf + cdata.nuelem_);
  set_shared_memory(threadIdx.x, sd, sf, si, cdata, size1, size2, size3, size4, size5);
  __syncthreads();

  for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
      double weight = get_weight_from_layer(size4, detid.layer(), sd);
      double rcorr = get_thickness_correction(size3, detid.type(), sd);
      double noise = get_noise(size2, detid.type(), sd);
      double cce_correction = get_cce_correction(size1, detid.type(), sd);
      double fCPerMIP = get_fCPerMIP(2, detid.type(), sd);
      double sigmaNoiseGeV = 1e-3 * weight * rcorr * __fdividef( noise,  fCPerMIP );
      make_rechit(i, dst_soa, src_soa, false, weight, rcorr, cce_correction, sigmaNoiseGeV, sf);
    }
}

__global__
void heb_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChebUncalibratedRecHitConstantData cdata, int length)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  HeterogeneousHGCScintillatorDetId detid(src_soa.id_[tid]);
  int size1 = cdata.s_weights_ + 3; 

  extern __shared__ double s[];
  double   *sd = s;
  float    *sf = (float*)   (sd + cdata.ndelem_);
  set_shared_memory(threadIdx.x, sd, sf, cdata, size1);
  __syncthreads();

  for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
      double weight = get_weight_from_layer(3, detid.layer(), sd);
      double noise = sd[2];
      double sigmaNoiseGeV = 1e-3 * noise * weight;
      make_rechit(i, dst_soa, src_soa, true, weight, 0., 0., sigmaNoiseGeV, sf);
    }
}
