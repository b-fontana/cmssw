#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "DataFormats/HGCRecHit/interface/HGCRecHit.h"
#include "DataFormats/ForwardDetId/interface/HGCalDetId.h"
#include "HGCalRecHitKernelImpl.cuh"

__device__ 
float get_weight_from_layer(const int& layer, const double (&weights)[maxsizes_constants::hef_weights])
{
  return (float)weights[layer];
}

__device__
void make_rechit_silicon(unsigned int tid, HGCRecHitSoA& dst_soa, HGCUncalibratedRecHitSoA& src_soa,
			 const float& weight, const float& rcorr, const float& cce_correction, const float &sigmaNoiseGeV,
			 const float& xmin, const float& xmax, const float& aterm, const float& cterm)
{
  dst_soa.id_[tid] = src_soa.id_[tid];
  dst_soa.energy_[tid] = src_soa.amplitude_[tid] * weight * 0.001f * __fdividef(rcorr, cce_correction);
  dst_soa.time_[tid] = src_soa.jitter_[tid];

  HeterogeneousHGCSiliconDetId detid(src_soa.id_[tid]);
  dst_soa.flagBits_[tid] = 0 | (0x1 << HGCRecHit::kGood);
  float son = __fdividef( dst_soa.energy_[tid], sigmaNoiseGeV);
  float son_norm = fminf(32.f, son) / 32.f * ((1 << 8)-1);
  long int son_round = lroundf( son_norm );
  //there is an extra 0.125 factor in HGCRecHit::signalOverSigmaNoise(), which should not affect CPU/GPU comparison
  dst_soa.son_[tid] = static_cast<uint8_t>( son_round );

  //get time resolution
  //https://github.com/cms-sw/cmssw/blob/master/RecoLocalCalo/HGCalRecProducers/src/ComputeClusterTime.cc#L50
  /*Maxmin trick to avoid conditions within the kernel (having xmin < xmax)
    3 possibilities: 1) xval -> xmin -> xmax
    2) xmin -> xval -> xmax
    3) xmin -> xmax -> xval
    The time error is calculated with the number in the middle.
  */
  float denominator = fminf( fmaxf(son, xmin), xmax);
  float div_ = __fdividef(aterm, denominator);
  dst_soa.timeError_[tid] = dst_soa.time_[tid] < 0 ? -1 : __fsqrt_rn( div_*div_ + cterm*cterm );
  //if dst_soa.time_[tid] < 1 always, then the above conditional expression can be replaced by
  //dst_soa.timeError_[tid] = fminf( fmaxf( dst_soa.time_[tid]-1, -1 ), sqrt( div_*div_ + cterm*cterm ) )
  //which is *not* conditional, and thus potentially faster; compare to HGCalRecHitWorkerSimple.cc
}

__device__
void make_rechit_scintillator(unsigned int tid, HGCRecHitSoA& dst_soa, HGCUncalibratedRecHitSoA& src_soa,
			      const float& weight, const float &sigmaNoiseGeV)
{
  dst_soa.id_[tid] = src_soa.id_[tid];
  dst_soa.energy_[tid] = src_soa.amplitude_[tid] * weight * 0.001f;
  dst_soa.time_[tid] = src_soa.jitter_[tid];

  HeterogeneousHGCScintillatorDetId detid(src_soa.id_[tid]);
  dst_soa.flagBits_[tid] = 0 | (0x1 << HGCRecHit::kGood);
  float son = __fdividef( dst_soa.energy_[tid], sigmaNoiseGeV);
  float son_norm = fminf(32.f, son) / 32.f * ((1 << 8)-1);
  long int son_round = lroundf( son_norm );
  //there is an extra 0.125 factor in HGCRecHit::signalOverSigmaNoise(), which should not affect CPU/GPU comparison
  dst_soa.son_[tid] = static_cast<uint8_t>( son_round );
  dst_soa.timeError_[tid] = -1;
}

__device__ 
float get_thickness_correction(const int& type, const double (&rcorr)[maxsizes_constants::hef_rcorr])
{
  return __fdividef( 1.f,  (float)rcorr[type] );
}

__device__
float get_noise(const int& type, const double (&noise_fC)[maxsizes_constants::hef_noise_fC])
{
  return (float)noise_fC[type];
}

__device__
float get_cce_correction(const int& type, const double (&cce)[maxsizes_constants::hef_cce])
{
  return (float)cce[type];
}

__device__ 
float get_fCPerMIP(const int& type, const double (&fCPerMIP)[maxsizes_constants::hef_fCPerMIP])
{
  return (float)fCPerMIP[type];
}

__global__
void ee_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGCeeUncalibratedRecHitConstantData cdata, int length)
{
}

__global__
void hef_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChefUncalibratedRecHitConstantData cdata, int length)
{
}

__global__
void heb_step1(HGCUncalibratedRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChebUncalibratedRecHitConstantData cdata, int length)
{
}

__global__
void ee_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGCeeUncalibratedRecHitConstantData cdata, int length)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
      HeterogeneousHGCSiliconDetId detid(src_soa.id_[i]);
      float weight         = get_weight_from_layer(detid.layer(), cdata.weights_);
      float rcorr          = get_thickness_correction(detid.type(), cdata.rcorr_);
      float noise          = get_noise(detid.type(), cdata.noise_fC_);
      float cce_correction = get_cce_correction(detid.type(), cdata.cce_);
      float fCPerMIP       = get_fCPerMIP(detid.type(), cdata.fCPerMIP_);
      float sigmaNoiseGeV  = 1e-3 * weight * rcorr * __fdividef( noise,  fCPerMIP );
      make_rechit_silicon(i, dst_soa, src_soa, weight, rcorr, cce_correction, sigmaNoiseGeV,
			  cdata.xmin_, cdata.xmax_, cdata.aterm_, cdata.cterm_);
    }
}

__global__
void hef_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChefUncalibratedRecHitConstantData cdata, const hgcal_conditions::HeterogeneousHEFConditionsESProduct* conds, int length)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  
  for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
      /*Uncomment the lines set to 1. as soon as those factors are centrally defined for the HSi.
	CUDADataFormats/HGCal/interface/HGCUncalibratedRecHitsToRecHitsConstants.h maxsizes_constanats will perhaps have to be changed (change some 3's to 6's) 
      */
      HeterogeneousHGCSiliconDetId detid(src_soa.id_[i]);
      uint32_t layer = detid.layer() + cdata.layerOffset_;
      float weight         = get_weight_from_layer(layer, cdata.weights_);
      float rcorr          = 1.f;//get_thickness_correction(detid.type(), cdata.rcorr_);
      float noise          = get_noise(detid.type(), cdata.noise_fC_);
      float cce_correction = 1.f;//get_cce_correction(detid.type(), cdata.cce_);
      float fCPerMIP       = get_fCPerMIP(detid.type(), cdata.fCPerMIP_);
      float sigmaNoiseGeV  = 1e-3 * weight * rcorr * __fdividef( noise,  fCPerMIP );
      make_rechit_silicon(i, dst_soa, src_soa, weight, rcorr, cce_correction, sigmaNoiseGeV,
			  cdata.xmin_, cdata.xmax_, cdata.aterm_, cdata.cterm_);
    }
}

__global__
void heb_to_rechit(HGCRecHitSoA dst_soa, HGCUncalibratedRecHitSoA src_soa, const HGChebUncalibratedRecHitConstantData cdata, int length)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x)
    {
      HeterogeneousHGCScintillatorDetId detid(src_soa.id_[i]);
      uint32_t layer = detid.layer() + cdata.layerOffset_;
      float weight        = get_weight_from_layer(layer, cdata.weights_);
      float noise         = cdata.noise_MIP_;
      float sigmaNoiseGeV = 1e-3 * noise * weight;
      make_rechit_scintillator(i, dst_soa, src_soa, weight, sigmaNoiseGeV);
    }
}

__global__
void fill_positions_from_detids(const hgcal_conditions::HeterogeneousHEFConditionsESProduct* conds)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < conds->nelems_posmap; i += blockDim.x * gridDim.x)
    {
      HeterogeneousHGCSiliconDetId did(conds->posmap.detid[i]);
      const float cU     = static_cast<float>( did.cellU()  );
      const float cV     = static_cast<float>( did.cellV()  );
      const float wU     = static_cast<float>( did.waferU() );
      const float wV     = static_cast<float>( did.waferV() );
      const float ncells = static_cast<float>( did.nCells() );
      const int32_t layer =  did.layer();
      
      //based on `std::pair<float, float> HGCalDDDConstants::locateCell(const HGCSiliconDetId&, bool)
      const float r_x2 = conds->posmap.waferSize + conds->posmap.sensorSeparation;
      const float r = 0.5f * r_x2;
      const float sqrt3 = __fsqrt_rn(3.f);
      const float rsqrt3 = __frsqrt_rn(3.f); //rsqrt: 1 / sqrt
      const float R = r_x2 * rsqrt3;
      const float n2 = ncells / 2.f;
      const float yoff_abs = rsqrt3 * r_x2;
      const float yoff = (layer%2==1) ? yoff_abs : -1.f * yoff_abs; //CHANGE according to Sunanda's reply
      float xpos = (-2.f * wU + wV) * r;
      float ypos = yoff + (1.5f * wV * R);
      const float R1 = __fdividef( conds->posmap.waferSize, 3.f * ncells );
      const float r1_x2 = R1 * sqrt3;
      xpos += (1.5f * (cV - ncells) + 1.f) * R1;
      ypos += (cU - 0.5f * cV - n2) * r1_x2;

      conds->posmap.x[i] = xpos; //* side; multiply by -1 if one wants to obtain the position from the opposite endcap. CAREFUL WITH LATER DETECTOR ALIGNMENT!!!
      conds->posmap.y[i] = ypos;

      //printf( "%d - %lf - %lf\n", cV - ncells, 1.5f*(static_cast<float>(cV) - static_cast<float>(ncells)), 1.5f*(static_cast<float>(cV - ncells)) );
      //printf("waferU: %d\t waferV: %d\t cellU: %d\t cellV: %d\t nCells: %d\t R1: %lf\t Layer: %d\t PosX: %lf\t PosY: %lf\t PosZ: %lf\n", wU, wV, cU, cV, ncells, R1, layer, conds->posmap.x[i], conds->posmap.y[i], conds->posmap.z[i]);
    }
}

__global__
void print_positions_from_detids(const hgcal_conditions::HeterogeneousHEFConditionsESProduct* conds)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < conds->nelems_posmap; i += blockDim.x * gridDim.x)
    {
      HeterogeneousHGCSiliconDetId did(conds->posmap.detid[i]);
      const int32_t layer = did.layer();
      float posz = conds->posmap.z_per_layer[ layer-1 ];
      printf("PosX: %lf\t PosY: %lf\t Posz: %lf\n", conds->posmap.x[i], conds->posmap.y[i], posz);
    } 
}
